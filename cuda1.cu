//including important libraries 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// definning n4
#define N 4

__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int i = threadIdx.x; //creating variables (int)
           int j = threadIdx.y; //creating variables (int)

           C[i][j] = A[i][j] + B[i][j];
}


int main(){//main function

  int A[N][N] =
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int B[N][N] = 
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int C[N][N] = 
     {
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0}
   };

  int (*d_A)[N], (*d_B)[N], (*d_C)[N];

  hipMalloc((void**)&d_A, (N*N)*sizeof(int));
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));

  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

  int numBlocks = 1;
  dim3 threadsPerBlock(N,N);
  MatAdd<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);

  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

  int i, j; printf("C = \n");
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");

  return 0;
}

