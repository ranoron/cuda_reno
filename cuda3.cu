
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 600

__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int i = blockIdx.x;// genarating random genarated multidiomentional arrays
           int j = blockIdx.y;

           C[i][j] = A[i][j] + B[i][j];  // genarating random genarated multidiomentional arrays
}

//int** randmatfunc();


void randmatfunc(int newmat[N][N]){
  int i, j, k; 
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
          k = rand() % 100 + 1;;        // printing those multidiomentional array list over here using for loop
            printf("%d ", k);
            newmat[i][j] =k;
        }
        printf("\n");
       
    } 
  printf("\n--------------------------------------\n");    //printing new line (multidiomentional arrays)
}

int main(){

int A[N][N];  
randmatfunc(A);   //inside the main function calling randumfunction (A) 
  
int B[N][N];  
randmatfunc(B);    //inside the main function calling randumfunction (B) 



  int C[N][N];

  int (*d_A)[N], (*d_B)[N], (*d_C)[N];   // calculating genaratedarrays 

  hipMalloc((void**)&d_A, (N*N)*sizeof(int));  // Allocates size bytes of linear memory on the device and returns in *devPtr a pointer to the allocated memory. returns cudaSuccess, cudaErrorMemoryAllocation
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));  // Allocates size bytes of linear memory on the device and returns in *devPtr a pointer to the allocated memory. returns cudaSuccess, cudaErrorMemoryAllocation
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));  // Allocates size bytes of linear memory on the device and returns in *devPtr a pointer to the allocated memory. returns cudaSuccess, cudaErrorMemoryAllocation

  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);  //Copies count bytes from the memory area pointed to by src to the memory area pointed to by dst, where kind is one of cudaMemcpyHostToHost, cudaMemcpyHostToDevice, 
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);  //Copies count bytes from the memory area pointed to by src to the memory area pointed to by dst, where kind is one of cudaMemcpyHostToHost, cudaMemcpyHostToDevice, 
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);  //Copies count bytes from the memory area pointed to by src to the memory area pointed to by dst, where kind is one of cudaMemcpyHostToHost, cudaMemcpyHostToDevice, 

  int numThreads = 1;
  dim3 numBlocks(N,N);
  MatAdd<<<numBlocks,numThreads>>>(d_A,d_B,d_C);

  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);   //CUDA memory copy types

  int i, j; printf("C = \n");
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", C[i][j]);    // printing new lines with printed data (multidiomentional array)
        }
        printf("\n");
    }

  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");

  return 0;
}

