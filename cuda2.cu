
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 22

__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int i = threadIdx.x; // create threds for use 1024 threads in a single block in a single dimension
           int j = threadIdx.y; // create threds for use 1024 threads in a single block in a single dimension

           C[i][j] = A[i][j] + B[i][j]; //calculation between arrays 
}

//int** randmatfunc();


void randmatfunc(int newmat[N][N]){   // genarating random genarated multidiomentional arrays
  int i, j, k; 
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
          k = rand() % 100 + 1;;
            printf("%d ", k);
            newmat[i][j] =k;      // printing those multidiomentional array list over here using for loop
        }
        printf("\n");
       
    } 
  printf("\n--------------------------------------\n");   //printing new line 
}

int main(){

int A[N][N];  
randmatfunc(A);     //inside the main function calling randumfunction (A) 
  
int B[N][N];  
randmatfunc(B);  //inside the main function calling randumfunction (B) 



  int C[N][N];

  int (*d_A)[N], (*d_B)[N], (*d_C)[N];  // calculating genarated multidiomentional arrays 

  hipMalloc((void**)&d_A, (N*N)*sizeof(int));  // Allocates size bytes of linear memory on the device and returns in *devPtr a pointer to the allocated memory. returns cudaSuccess, cudaErrorMemoryAllocation
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));// Allocates size bytes of linear memory on the device and returns in *devPtr a pointer to the allocated memory. returns cudaSuccess, cudaErrorMemoryAllocation
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));// Allocates size bytes of linear memory on the device and returns in *devPtr a pointer to the allocated memory. returns cudaSuccess, cudaErrorMemoryAllocation

  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice); //Copies count bytes from the memory area pointed to by src to the memory area pointed to by dst, where kind is one of cudaMemcpyHostToHost, cudaMemcpyHostToDevice, 
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);//Copies count bytes from the memory area pointed to by src to the memory area pointed to by dst, where kind is one of cudaMemcpyHostToHost, cudaMemcpyHostToDevice, 
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);//Copies count bytes from the memory area pointed to by src to the memory area pointed to by dst, where kind is one of cudaMemcpyHostToHost, cudaMemcpyHostToDevice, 

  int numBlocks = 1;
  dim3 threadsPerBlock(N,N);  // is an integer struct type defined in the file
  MatAdd<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);

  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);  //CUDA memory copy types

  int i, j; printf("C = \n");
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", C[i][j]);
        }
        printf("\n");  // printing new lines (multidiomentional arrays)
    }

  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");

  return 0;
}

